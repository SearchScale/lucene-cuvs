#include "com_searchscale_lucene_vectorsearch_jni_CuVSIndexJni.h"
#include <cstdint>
#include <sys/time.h>
#include <vector>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/resources.hpp>
#include <raft/neighbors/cagra.cuh>
#include <raft/neighbors/cagra_serialize.cuh>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

long ms () {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return tp.tv_sec * 1000 + tp.tv_usec / 1000; // get current timestamp in milliseconds
}

raft::neighbors::cagra::index_params index_params;
raft::neighbors::cagra::search_params search_params;
raft::device_resources dev_resources;
rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(rmm::mr::get_current_device_resource(), 2 * 1024 * 1024 * 1024ull);
std::string filename("./cagra_index.indx");
raft::neighbors::cagra::index<float, uint32_t> dindx = raft::neighbors::cagra::index<float, uint32_t>(dev_resources);

JNIEXPORT jint JNICALL Java_com_searchscale_lucene_vectorsearch_jni_CuVSIndexJni_initIndex
(JNIEnv *env, jobject jobj, jintArray docIds, jfloatArray dataVectors, jint numVectors, jint dimension) {
  std::cout<<"CUDA devices: "<<rmm::get_num_cuda_devices()<<std::endl;
  rmm::mr::set_current_device_resource(&pool_mr);

  // Copy the arrays from JNI to local variables.
  // TODO: Instead of copying three times (JNI->array->hostmatrix->devicematrix),
  // TODO: it might possible to do it once (JNI -> Device) for better efficiency.
  long startTime = ms();
  jsize numDocs = env->GetArrayLength(docIds);
  std::vector<int> docs (numDocs);
  env->GetIntArrayRegion( docIds, 0, numDocs, &docs[0] ); // TODO: This docid to index mapping should be persisted and used during search
  std::vector<float> data(numVectors * dimension);
  env->GetFloatArrayRegion( dataVectors, 0, numVectors * dimension, &data[0] );
  auto datasetHost = raft::make_host_matrix<float, int64_t>(dev_resources, numVectors, dimension);
  auto dataset = raft::make_device_matrix<float, int64_t>(dev_resources, numVectors, dimension);
  int p = 0;
  for(size_t i = 0; i < numDocs ; i ++) {
      for(size_t j = 0; j < dimension; ++j) {
          datasetHost(i, j) = data[p++]; // TODO: Is there a better SIMD friendly way to copy?
      }
  }
  hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);
  raft::copy(dataset.data_handle(), datasetHost.data_handle(), datasetHost.size(), stream);
  raft::resource::sync_stream(dev_resources, stream);
  std::cout<<"Data copying time (CPU to GPU): "<<(ms()-startTime)<<std::endl;

  // Build the index
  startTime = ms();
  auto ind = raft::neighbors::cagra::build<float, uint32_t>(dev_resources, index_params, raft::make_const_mdspan(dataset.view()));
  std::cout << "Cagra Index building time: " << (ms()-startTime) << std::endl;

  // Serialize the index into a file
  raft::neighbors::cagra::serialize(dev_resources, filename, ind);
  dindx = raft::neighbors::cagra::deserialize<float, uint32_t>(dev_resources, filename);
  return numVectors * dimension;
}

JNIEXPORT jobject JNICALL Java_com_searchscale_lucene_vectorsearch_jni_CuVSIndexJni_getTopK
(JNIEnv *env, jobject jobj, jfloatArray queryVector, jint topK)
{
  rmm::mr::set_current_device_resource(&pool_mr);

  // Copy the query vector into the device
  int64_t topk = topK;
  int64_t n_queries = 1;
  jsize queryVectorSize = env->GetArrayLength(queryVector);
  std::vector<float> query(queryVectorSize);
  env->GetFloatArrayRegion( queryVector, 0, queryVectorSize, &query[0] );
  auto queries = raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, queryVectorSize); // one query at a time
  for (int i = 0; i < queryVectorSize; i++) {
    queries(0, i) = query[i];
  }

  // Perform the search
  long startTime = ms();
  auto neighbors = raft::make_device_matrix<uint32_t>(dev_resources, n_queries, topk);
  auto distances = raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, topk);
  raft::neighbors::cagra::search<float, uint32_t>(dev_resources, search_params, dindx, raft::make_const_mdspan(queries.view()), neighbors.view(), distances.view());
  std::cout<<"Time taken for cagra::search: "<<(ms()-startTime)<<std::endl;

  // Return the results (neighbors and distances)
  int numResults = distances.extent(1);
  float *retDocsAndScores = (float*)malloc( sizeof(int)*numResults + sizeof(float)*numResults );
  int               *docs = &((int*)retDocsAndScores)[numResults];
  for (int i=0; i<numResults; i++) { // TODO: Is there a better SIMD friendly copy (like thrust::copy)?
    docs[i] = neighbors(0, i);
    retDocsAndScores[i] = distances(0, i);
  }
  jobject directBuffer = env->NewDirectByteBuffer((void*)retDocsAndScores, sizeof(int)*numResults + sizeof(float)*numResults);
  return directBuffer;
}
